#include "hip/hip_runtime.h"
#include <vector>
#include <cassert>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <set>

#include "timer.cpp"

using namespace std;

typedef uint64_t u64;
typedef int64_t i64;

#define GPU_ERROR_CHECK(x) { gpuAssert((x), __FILE__, __LINE__); }
#define _cudaMalloc(...) GPU_ERROR_CHECK(hipMalloc(__VA_ARGS__));
#define _cudaMemcpy(...) GPU_ERROR_CHECK(hipMemcpy(__VA_ARGS__));
#define _cudaDeviceSynchronize(...) GPU_ERROR_CHECK(hipDeviceSynchronize(__VA_ARGS__));

__host__ inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
	if (code == hipSuccess) return;
	fprintf(stderr, "GPU Assert: %s %s %d\n", hipGetErrorString(code), file, line);
	if (abort) exit(code);
}

class AdjacencyGraph {
	public:
		u64 n;
		u64 m;
		u64* offsets;
		u64* edges;
		AdjacencyGraph(u64 n, u64 m) : n(n), m(m), offsets{new u64[n]}, edges{new u64[m]} {}
};

__host__ AdjacencyGraph load_adjacency_graph(const string& path) {
	auto file = ifstream(path);

	string t;
	u64 n, m;

	file >> t >> n >> m;

	assert(t == "AdjacencyGraph");

	auto graph = AdjacencyGraph(n, m);

	for (auto i = 0; i < n; i++) file >> graph.offsets[i];
	for (auto i = 0; i < m; i++) file >> graph.edges[i];

	return graph;
}

class BFSJob {
	public:
	// Generic
	bool* frontier;
	bool xdone = false;
	bool done = false;
	// BFS-Specific
	u64 root;
	i64* parents;

	BFSJob(u64 root) : root(root) {}

	__host__ void init(AdjacencyGraph& graph) {
		bool* h_frontier = new bool[graph.n];
		i64* h_parents = new i64[graph.n];
		for (u64 i = 0; i < graph.n; i++) {
			h_frontier[i] = i == root;
			h_parents[i] = i == root ? (i64) root : (i64) -1;
		}

		bool* d_frontier;
		i64* d_parents;
		auto x = graph.n * sizeof(bool);
		auto y = graph.n * sizeof(i64);
		_cudaMalloc(&d_frontier, x);
		_cudaMemcpy(d_frontier, h_frontier, x, hipMemcpyHostToDevice);
		_cudaMalloc(&d_parents, y);
		_cudaMemcpy(d_parents, h_parents, y, hipMemcpyHostToDevice);

		frontier = d_frontier;
		parents = d_parents;
	}

	__device__ void iter(u64* d_offsets, u64* d_edges, u64 n, u64 m, u64 id) {
		u64 a = d_offsets[id];
		u64 b = id < n - 1 ? d_offsets[id + 1] : n;

		frontier[id] = false;

		for (auto i = a; i < b; i++) {
			u64 c = d_edges[i];
			if (parents[c] != -1) continue;
			parents[c] = id;
			frontier[c] = true;
			done = false;
		}
	}
};

__global__ void iter(u64* d_offsets, u64* d_edges, u64 n, u64 m, BFSJob* jobs, u64 j) {
	// Get our global thread ID
	u64 id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= n) return;

	for (auto i = 0; i < j; i++) {
		if (jobs[i].xdone) continue;
		if (!jobs[i].frontier[id]) continue;
		jobs[i].iter(d_offsets, d_edges, n, m, id);
	}
}

void exec(int count, AdjacencyGraph& graph, u64* d_offsets, u64* d_edges, int offset) {
	BFSJob* h_jobs = (BFSJob*) malloc(count * sizeof(BFSJob));
	BFSJob* d_jobs;
	for (auto i = 0; i < count; i++) {
		auto bfs = BFSJob((i + 1 + offset) * 10);
		bfs.init(graph);
		h_jobs[i] = bfs;
	}
	_cudaMalloc(&d_jobs, count * sizeof(BFSJob));
	_cudaMemcpy(d_jobs, h_jobs, count * sizeof(BFSJob), hipMemcpyHostToDevice);

	// Number of threads in each thread block
	u64 blockSize = 1024;
 
	// Number of thread blocks in grid
	u64 gridSize = (u64) ceil((float) graph.n / blockSize);

	while (true) {
		auto done = true;
		bool* j_done = (bool*) malloc(sizeof(bool));
		for (auto i = 0; i < count; i++) {
			_cudaMemcpy(j_done, &(d_jobs[i].done), sizeof(bool), hipMemcpyDeviceToHost);
			if (!(*j_done)) {
				done = false;
				*j_done = true;
				_cudaMemcpy(&(d_jobs[i].done), j_done, sizeof(bool), hipMemcpyHostToDevice);
			} else {
				*j_done = true;
				_cudaMemcpy(&(d_jobs[i].xdone), j_done, sizeof(bool), hipMemcpyHostToDevice);
			}
		}
		if (done) break;
		iter<<<gridSize, blockSize>>>(d_offsets, d_edges, graph.m, graph.n, d_jobs, count);
		_cudaDeviceSynchronize();
	}
}

__host__ int main(int argc, char **argv) {
	auto graph_path = string(argv[1]);
	auto job_count = (u64) atoi(argv[2]);

	cout << "Graph Path: " << graph_path << endl;
	cout << "Job Count: " << job_count << endl;

	auto host_io_time = custom::Timer("Host IO");

	auto graph = load_adjacency_graph(graph_path);
	
	host_io_time.report();

	auto device_io_time = custom::Timer("Device IO");

	// TODO: Create type for offset and another type for edge
	u64* d_offsets;
	u64* d_edges;

	auto offsets_size = graph.n * sizeof(u64);
	auto edges_size = graph.m * sizeof(u64);
 
	// Allocate memory for each vector on GPU
	_cudaMalloc(&d_offsets, offsets_size);
	_cudaMalloc(&d_edges, edges_size);

	// Copy host vectors to device
	_cudaMemcpy(d_offsets, graph.offsets, offsets_size, hipMemcpyHostToDevice);
	_cudaMemcpy(d_edges, graph.edges, edges_size, hipMemcpyHostToDevice);

	device_io_time.report();

	auto separated_jobs_time = custom::Timer("Running separated jobs");
	for (auto i = 0; i < job_count; i++) {
		exec(1, graph, d_offsets, d_edges, i);
	}
	separated_jobs_time.report();

	cout << "Running one job took " << (separated_jobs_time.seconds() / job_count) << " seconds." << std::endl;

	auto merged_jobs_time = custom::Timer("Running merged jobs");
	exec(job_count, graph, d_offsets, d_edges, 0);
	merged_jobs_time.report();
}

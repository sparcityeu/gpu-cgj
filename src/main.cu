#include "hip/hip_runtime.h"
#include <vector>
#include <cassert>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <set>

#include "timer.cpp"

using namespace std;

typedef uint64_t u64;
typedef int64_t i64;

#define GPU_ERROR_CHECK(x) { gpuAssert((x), __FILE__, __LINE__); }
#define _cudaMalloc(...) GPU_ERROR_CHECK(hipMalloc(__VA_ARGS__));
#define _cudaMemcpy(...) GPU_ERROR_CHECK(hipMemcpy(__VA_ARGS__));
#define _cudaDeviceSynchronize(...) GPU_ERROR_CHECK(hipDeviceSynchronize(__VA_ARGS__));

__host__ inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
	if (code == hipSuccess) return;
	fprintf(stderr, "GPU Assert: %s %s %d\n", hipGetErrorString(code), file, line);
	if (abort) exit(code);
}

class AdjacencyGraph {
	public:
		u64 n;
		u64 m;
		u64* offsets;
		u64* edges;
		AdjacencyGraph(u64 n, u64 m) : n(n), m(m), offsets{new u64[n]}, edges{new u64[m]} {}
};

__host__ AdjacencyGraph load_adjacency_graph(const string& path) {
	auto file = ifstream(path);

	string t;
	u64 n, m;

	file >> t >> n >> m;

	assert(t == "AdjacencyGraph");

	auto graph = AdjacencyGraph(n, m);

	for (auto i = 0; i < n; i++) file >> graph.offsets[i];
	for (auto i = 0; i < m; i++) file >> graph.edges[i];

	return graph;
}

class BFSJob {
	public:
	// Generic
	bool* frontier;
	bool xdone = false;
	bool done = false;
	// BFS-Specific
	u64 root;
	i64* parents;

	BFSJob(u64 root) : root(root) {}

	__host__ void init(AdjacencyGraph& graph) {
		bool* h_frontier = new bool[graph.n];
		i64* h_parents = new i64[graph.n];
		for (u64 i = 0; i < graph.n; i++) {
			h_frontier[i] = i == root;
			h_parents[i] = i == root ? (i64) root : (i64) -1;
		}

		bool* d_frontier;
		i64* d_parents;
		auto x = graph.n * sizeof(bool);
		auto y = graph.n * sizeof(i64);
		_cudaMalloc(&d_frontier, x);
		_cudaMemcpy(d_frontier, h_frontier, x, hipMemcpyHostToDevice);
		_cudaMalloc(&d_parents, y);
		_cudaMemcpy(d_parents, h_parents, y, hipMemcpyHostToDevice);

		frontier = d_frontier;
		parents = d_parents;
	}

	__device__ void iter(u64* d_offsets, u64* d_edges, u64 n, u64 m, u64 id) {
		u64 a = d_offsets[id];
		u64 b = id < n - 1 ? d_offsets[id + 1] : n;

		frontier[id] = false;

		for (auto i = a; i < b; i++) {
			u64 c = d_edges[i];
			if (parents[c] != -1) continue;
			parents[c] = id;
			frontier[c] = true;
			done = false;
		}
	}
};

class SSSPJob {
	public:
	u64 source;
	i64* dists;

	SSSPJob(u64 source) : source(source) {}

	__host__ void init(AdjacencyGraph& graph) {
		i64* h_dists = new i64[graph.n];
		for (u64 i = 0; i < graph.n; i++) {
			h_dists[i] = i == source ? (i64) 0 : (i64) INT_FAST64_MAX;
		}

		i64* d_dists;
		auto y = graph.n * sizeof(i64);
		_cudaMalloc(&d_dists, y);
		_cudaMemcpy(d_dists, h_dists, y, hipMemcpyHostToDevice);

		dists = d_dists;
	}

	__device__ void iter(u64* d_offsets, u64* d_edges, u64 n, u64 m, u64 id) {
		u64 a = d_offsets[id];
		u64 b = id < n - 1 ? d_offsets[id + 1] : n;

		for (auto i = a; i < b; i++) {
			auto dst = d_edges[i];
			if (dists[id] + 1 < dists[dst]) dists[dst] = dists[id] + 1;
		}
	}
};

__global__ void iter_bfs(u64* d_offsets, u64* d_edges, u64 n, u64 m, BFSJob* jobs, u64 j) {
	// Get our global thread ID
	u64 id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= n) return;

	for (auto i = 0; i < j; i++) {
		if (jobs[i].xdone) continue;
		if (!jobs[i].frontier[id]) continue;
		jobs[i].iter(d_offsets, d_edges, n, m, id);
	}
}

__global__ void iter_sssp(u64* d_offsets, u64* d_edges, u64 n, u64 m, SSSPJob* jobs, u64 j) {
	// Get our global thread ID
	u64 id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= n) return;

	for (auto i = 0; i < j; i++) {
		jobs[i].iter(d_offsets, d_edges, n, m, id);
	}
}

void exec_bfs(int count, AdjacencyGraph& graph, u64* d_offsets, u64* d_edges, int offset) {
	BFSJob* h_jobs = (BFSJob*) malloc(count * sizeof(BFSJob));
	BFSJob* d_jobs;
	for (auto i = 0; i < count; i++) {
		auto bfs = BFSJob((i + 1 + offset) * 10);
		bfs.init(graph);
		h_jobs[i] = bfs;
	}
	_cudaMalloc(&d_jobs, count * sizeof(BFSJob));
	_cudaMemcpy(d_jobs, h_jobs, count * sizeof(BFSJob), hipMemcpyHostToDevice);

	// Number of threads in each thread block
	u64 blockSize = 1024;
 
	// Number of thread blocks in grid
	u64 gridSize = (u64) ceil((float) graph.n / blockSize);

	while (true) {
		auto done = true;
		bool* j_done = (bool*) malloc(sizeof(bool));
		for (auto i = 0; i < count; i++) {
			_cudaMemcpy(j_done, &(d_jobs[i].done), sizeof(bool), hipMemcpyDeviceToHost);
			if (!(*j_done)) {
				done = false;
				*j_done = true;
				_cudaMemcpy(&(d_jobs[i].done), j_done, sizeof(bool), hipMemcpyHostToDevice);
			} else {
				*j_done = true;
				_cudaMemcpy(&(d_jobs[i].xdone), j_done, sizeof(bool), hipMemcpyHostToDevice);
			}
		}
		if (done) break;
		iter_bfs<<<gridSize, blockSize>>>(d_offsets, d_edges, graph.m, graph.n, d_jobs, count);
		_cudaDeviceSynchronize();
	}
}

void exec_sssp(int count, AdjacencyGraph& graph, u64* d_offsets, u64* d_edges, int offset) {
	SSSPJob* h_jobs = (SSSPJob*) malloc(count * sizeof(SSSPJob));
	SSSPJob* d_jobs;
	for (auto i = 0; i < count; i++) {
		auto sssp = SSSPJob((i + 1 + offset) * 10);
		sssp.init(graph);
		h_jobs[i] = sssp;
	}
	_cudaMalloc(&d_jobs, count * sizeof(SSSPJob));
	_cudaMemcpy(d_jobs, h_jobs, count * sizeof(SSSPJob), hipMemcpyHostToDevice);

	// Number of threads in each thread block
	u64 blockSize = 1024;
 
	// Number of thread blocks in grid
	u64 gridSize = (u64) ceil((float) graph.n / blockSize);

	for (auto i = 0; i < 50; i++) {
		iter_sssp<<<gridSize, blockSize>>>(d_offsets, d_edges, graph.m, graph.n, d_jobs, count);
		_cudaDeviceSynchronize();
	}
}

__host__ int main(int argc, char **argv) {
	auto graph_path = string(argv[1]);
	auto job_count = (u64) atoi(argv[2]);
	auto run_bfs = (atoi(argv[3]) != 0);

	cout << "Graph Path: " << graph_path << endl;
	cout << "Job Count: " << job_count << endl;
	cout << "Job Type: " << (run_bfs ? "BFS" : "SSSP") << endl;

	auto host_io_time = custom::Timer("Host IO");

	auto graph = load_adjacency_graph(graph_path);
	
	host_io_time.report();

	auto device_io_time = custom::Timer("Device IO");

	// TODO: Create type for offset and another type for edge
	u64* d_offsets;
	u64* d_edges;

	auto offsets_size = graph.n * sizeof(u64);
	auto edges_size = graph.m * sizeof(u64);
 
	// Allocate memory for each vector on GPU
	_cudaMalloc(&d_offsets, offsets_size);
	_cudaMalloc(&d_edges, edges_size);

	// Copy host vectors to device
	_cudaMemcpy(d_offsets, graph.offsets, offsets_size, hipMemcpyHostToDevice);
	_cudaMemcpy(d_edges, graph.edges, edges_size, hipMemcpyHostToDevice);

	device_io_time.report();

	auto separated_jobs_time = custom::Timer("Running separated jobs");
	for (auto i = 0; i < job_count; i++) {
		if (run_bfs) exec_bfs(1, graph, d_offsets, d_edges, i);
		else exec_sssp(1, graph, d_offsets, d_edges, i);
	}
	separated_jobs_time.report();

	cout << "Running one job took " << (separated_jobs_time.seconds() / job_count) << " seconds." << std::endl;

	auto merged_jobs_time = custom::Timer("Running merged jobs");
	if (run_bfs) exec_bfs(job_count, graph, d_offsets, d_edges, 0);
	else exec_sssp(job_count, graph, d_offsets, d_edges, 0);
	merged_jobs_time.report();

	auto sep = separated_jobs_time.seconds();
	auto one = (sep / job_count);
	auto mer = merged_jobs_time.seconds();

	cout << one << "," << sep << "," << mer << endl;
}
